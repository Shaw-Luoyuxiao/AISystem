
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
// __global__ 变量声明符，作用是将 add 函数变成可以在 GPU 上运行的函数
// __global__ 函数被称为 Kernel
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<25;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  // 内存分配，在 GPU 或者 CPU 上统一分配内存
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run Kernel on 1M elements on the GPU
  // execution configuration, 执行配置
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  // CPU 需要等待 cuda 上的代码运行完毕，才能对数据进行读取
  hipDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}